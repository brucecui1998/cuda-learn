#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <chrono>
#include <iostream>

__global__
void saxpy(int n, float a, float* x, float* y)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) y[i] = a * x[i] + y[i];
}

int main() {
    int nDevices;

    hipError_t err = hipGetDeviceCount(&nDevices);
    if (err != hipSuccess) printf("%s\n", hipGetErrorString(err));

    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (KHz): %d\n",
            prop.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n",
            prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
            2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6);
    }

    int N = 1 << 20;
    float* x, * y, * d_x, * d_y;
    x = (float*)malloc(N * sizeof(float));
    y = (float*)malloc(N * sizeof(float));

    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_y, N * sizeof(float));

    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }


    hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);

    //Perform SAXPY on 1M elements
   /*float t1 = myCPUTimer();
   saxpy << <(N + 255) / 256, 256 >> > (N, 2.0f, d_x, d_y);
   hipDeviceSynchronize();
   float t2 = myCPUTimer();
   std::cout << "Kernel execution time: " << t2 - t1 << " ms" << std::endl;*/


    // Perform SAXPY on 1M elements
    saxpy << <(N + 255) / 256, 1025 >> > (N, 2.0, d_x, d_y);
    hipError_t errSync = hipGetLastError();
    hipError_t errAsync = hipDeviceSynchronize();
    if (errSync != hipSuccess)
        printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
    if (errAsync != hipSuccess)
        printf("Async kernel error: %s\n", hipGetErrorString(errAsync));

    hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);

    float milliseconds = 0;


    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, abs(y[i] - 4.0f));
    printf("Max error: %f\n", maxError);

    hipFree(d_x);
    hipFree(d_y);
    free(x);
    free(y);
}