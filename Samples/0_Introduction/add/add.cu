#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <math.h>
 //Kernel function to add the elements of two arrays
__global__
void add(int n, float* x, float* y)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
    {
        y[i] = x[i] + y[i];
        //printf("Thread %d in Block %d has index %d\n", threadIdx.x, blockIdx.x, index);
        if (i == 0) { // 仅在第一个线程中打印，避免重复输出
            printf("Block index: (%d, %d, %d), Thread index: (%d, %d, %d)\n",
                blockIdx.x, blockIdx.y, blockIdx.z,
                threadIdx.x, threadIdx.y, threadIdx.z);
            printf("Block dimensions: (%d, %d, %d), Grid dimensions: (%d, %d, %d)\n",
                blockDim.x, blockDim.y, blockDim.z,
                gridDim.x, gridDim.y, gridDim.z);
        }
    }
        
}

int main(void)
{
    int N = 1 << 20;
    float* x, * y;

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Run kernel on 1M elements on the GPU
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    add << <numBlocks, blockSize >> > (N, x, y);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);

    return 0;
}
